
#include <hip/hip_runtime.h>
// many tests depend on this file, don't change/move it unless you know what you are doing

extern "C" __global__ void sub(
    const int*   x,
    const int*   y,
          int*   z,
          int  len
) {
    for (
        int i = blockDim.x * blockIdx.x + threadIdx.x;
            i < len;
            i += gridDim.x * blockDim.x
    ) {
        z[i] = x[i] - y[i];
    }
}

